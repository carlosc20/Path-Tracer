#include "hip/hip_runtime.h"
#include "common.h"

// -------------------------------------------------------
// Lambert
extern "C" __global__ void __closesthit__radiance() {

    const TriangleMeshSBTData &sbtData
      = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();  

    RadiancePRD &prd = *(RadiancePRD *)getPRD<RadiancePRD>();

    // retrieve primitive id and indexes
    const int   primID = optixGetPrimitiveIndex();
    const uint3 index  = sbtData.index[primID];

    // get barycentric coordinates
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    // compute normal
    const float4 n
        = (1.f-u-v) * sbtData.vertexD.normal[index.x]
        +         u * sbtData.vertexD.normal[index.y]
        +         v * sbtData.vertexD.normal[index.z];

    float3 nn = normalize(make_float3(n));

    // intersection position
    const float3 &rayDir =  optixGetWorldRayDirection();
    const float3 pos = optixGetWorldRayOrigin() + optixGetRayTmax() * rayDir ;

    // invert normal if hit from behind
    if (dot(nn, rayDir) > 0.0)
        nn = -nn;

    // adds emission if first surface hit is a light or a bounce from a specular surface
    if ((prd.countEmitted || prd.specularBounce) && length(sbtData.emission) != 0) {
        prd.emitted = sbtData.emission;
        prd.done = true;
        return;
    }
    prd.emitted = make_float3(0.0f);
    prd.countEmitted = false;
    prd.specularBounce = false;

    uint32_t seed = prd.seed;

    {
        // set origin and direction for next ray

        const float z1 = rnd(seed);
        const float z2 = rnd(seed);

        float3 w_in;
        cosine_sample_hemisphere( z1, z2, w_in );
        Onb onb( nn );
        onb.inverse_transform( w_in );

        prd.direction = w_in;
        prd.origin    = pos;
    }
    

    const float z1 = rnd(seed);
    const float z2 = rnd(seed);
    prd.seed = seed;

    // random point from light area
    // square area 0.47 x 0.38, XZ plane
    const float3 lightV1 = make_float3(0.47f, 0.0, 0.0f);
    const float3 lightV2 = make_float3(0.0f, 0.0, 0.38f);
    const float3 light_pos = make_float3(optixLaunchParams.global->lightPos) + lightV1 * z1 + lightV2 * z2;

    // Calculate properties of light sample (for area based pdf)
    const float  Ldist = length( light_pos - pos );
    const float3 L     = normalize( light_pos - pos );
    const float  nDl   = dot( nn, L );
    const float3 Ln    = normalize(cross(lightV1, lightV2));
    const float  LnDl  = -dot( Ln, L );

    // check light sample occlusion
    float weight = 0.0f;
    if( nDl > 0.0f && LnDl > 0.0f ) {
        uint32_t occluded = 0u;
        optixTrace(optixLaunchParams.traversable,
            pos,
            L,
            0.1f,                    // tmin
            Ldist - 0.01f,           // tmax
            0.0f,                    // rayTime
            OptixVisibilityMask( 1 ),
            OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT,
            SHADOW,                 // SBT offset
            RAY_TYPE_COUNT,         // SBT stride
            SHADOW,                 // missSBTIndex
            occluded);

        if(!occluded) {
            const float att = Ldist * Ldist;
            const float A = length(cross(lightV1, lightV2));
            weight = nDl * LnDl * A  / att;
        }
    }

    const float3 Lintensity = make_float3(5.0f, 5.0f, 5.0f);

    prd.radiance += Lintensity * weight * optixLaunchParams.global->lightScale;

    if (sbtData.hasTexture && sbtData.vertexD.texCoord0) {  
        // compute pixel texture coordinate
        const float4 tc
          = (1.f-u-v) * sbtData.vertexD.texCoord0[index.x]
          +         u * sbtData.vertexD.texCoord0[index.y]
          +         v * sbtData.vertexD.texCoord0[index.z];
        // fetch texture value
        float4 fromTexture = tex2D<float4>(sbtData.texture,tc.x,tc.y);
        prd.attenuation *= make_float3(fromTexture);
    }
    else
        prd.attenuation *= sbtData.diffuse;

}


extern "C" __global__ void __anyhit__radiance() {}

extern "C" __global__ void __miss__radiance() {
    // miss sets the background color
    RadiancePRD &prd = *(RadiancePRD*)getPRD<RadiancePRD>();
    prd.radiance = make_float3(0.0f, 0.0f, 0.0f); // black
    prd.done = true;
}


// -----------------------------------------------
// Shadow rays

extern "C" __global__ void __closesthit__shadow() {
    optixSetPayload_0( static_cast<uint32_t>(true));
}

extern "C" __global__ void __anyhit__shadow() {}

extern "C" __global__ void __miss__shadow() {
    optixSetPayload_0( static_cast<uint32_t>(false));
}