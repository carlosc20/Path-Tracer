#include "hip/hip_runtime.h"
#include "optixParams.h" // our launch params


extern "C" {
    __constant__ LaunchParams optixLaunchParams;
}

// ray types
enum { RAIDANCE=0, SHADOW, RAY_TYPE_COUNT };

struct RadiancePRD{
    float3   emitted;
    float3   radiance;
    float3   attenuation;
    float3   origin;
    float3   direction;
    bool done;
    uint32_t seed;
    int32_t  countEmitted;
} ;

struct shadowPRD{
    float shadowAtt;
    uint32_t seed;
} ;




// -------------------------------------------------------

extern "C" __global__ void __closesthit__radiance() {

    const TriangleMeshSBTData &sbtData
      = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();  

    RadiancePRD &prd = *(RadiancePRD *)getPRD<RadiancePRD>();

    // retrieve primitive id and indexes
    const int   primID = optixGetPrimitiveIndex();
    const uint3 index  = sbtData.index[primID];

    // get barycentric coordinates
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    // compute normal
    const float4 n
        = (1.f-u-v) * sbtData.vertexD.normal[index.x]
        +         u * sbtData.vertexD.normal[index.y]
        +         v * sbtData.vertexD.normal[index.z];

    float3 nn = normalize(make_float3(n));

    // intersection position
    const float3 &rayDir =  optixGetWorldRayDirection();
    const float3 pos = optixGetWorldRayOrigin() + optixGetRayTmax() * rayDir ;

    if (dot(nn, rayDir) > 0.0)
        nn = -nn;


    // check if the ray hit a light and is first ray
    if (prd.countEmitted && length(sbtData.emission) != 0) {
        prd.emitted = sbtData.emission ;
        return;
    }
    else
        prd.emitted = make_float3(0.0f);

    uint32_t seed = prd.seed;

    {
        // trace ray to check if light is occluded
        const float z1 = rnd(seed);
        const float z2 = rnd(seed);

        float3 w_in;
        cosine_sample_hemisphere( z1, z2, w_in );
        Onb onb( nn );
        onb.inverse_transform( w_in );
        prd.direction = w_in;
        prd.origin    = pos;

        prd.attenuation *= sbtData.diffuse ;
        prd.countEmitted = false;
    }
    

    const float z1 = rnd(seed);
    const float z2 = rnd(seed);
    prd.seed = seed;

    // random point from light area
    const float3 lightV1 = make_float3(0.47f, 0.0, 0.0f);
    const float3 lightV2 = make_float3(0.0f, 0.0, 0.38f);
    const float3 light_pos = make_float3(optixLaunchParams.global->lightPos) + lightV1 * z1 + lightV2 * z2;

    // Calculate properties of light sample (for area based pdf)
    const float  Ldist = length(light_pos - pos );
    const float3 L     = normalize(light_pos - pos );
    const float  nDl   = dot( nn, L );
    const float3 Ln    = normalize(cross(lightV1, lightV2));
    const float  LnDl  = -dot( Ln, L );

    float weight = 0.0f;
    if( nDl > 0.0f && LnDl > 0.0f )
    {
        uint32_t occluded = 0u;
        optixTrace(optixLaunchParams.traversable,
            pos,
            L,
            0.1f,         // tmin
            Ldist - 0.01f,  // tmax
            0.0f,                    // rayTime
            OptixVisibilityMask( 1 ),
            OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT,
            SHADOW,      // SBT offset
            RAY_TYPE_COUNT,          // SBT stride
            SHADOW,      // missSBTIndex
            occluded);

        if( !occluded )
        {
            const float att = Ldist * Ldist;
            const float A = length(cross(lightV1, lightV2));
            weight = nDl * LnDl * A  / att;
        }
    }

    prd.radiance += make_float3(5.0f, 5.0f, 5.0f) * weight * optixLaunchParams.global->lightScale;

    /*
    https://computergraphics.stackexchange.com/questions/2316/is-russian-roulette-really-the-answer
    
    // Russian Roulette
        // Randomly terminate a path with a probability inversely equal to the throughput
        float p = std::max(throughput.x, std::max(throughput.y, throughput.z));
        if (sampler->NextFloat() > p) {
            break;
        }

        // Add the energy we 'lose' by randomly terminating paths
        throughput *= 1 / p;
    */
}


extern "C" __global__ void __anyhit__radiance() {

}


// miss sets the background color
extern "C" __global__ void __miss__radiance() {

    RadiancePRD &prd = *(RadiancePRD*)getPRD<RadiancePRD>();
    // set black as background color
    prd.radiance = make_float3(0.0f, 0.0f, 0.0f);
    prd.done = true;
}


// -----------------------------------------------
// Shadow rays

extern "C" __global__ void __closesthit__shadow() {

    optixSetPayload_0( static_cast<uint32_t>(true));
}


// any hit for shadows
extern "C" __global__ void __anyhit__shadow() {

}


// miss for shadows
extern "C" __global__ void __miss__shadow() {

    optixSetPayload_0( static_cast<uint32_t>(false));
}




// -----------------------------------------------
// Metal Phong rays (Materiais especulares com rugosidade)

extern "C" __global__ void __closesthit__phong_metal() {

    const TriangleMeshSBTData &sbtData
      = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();  

    // retrieve primitive id and indexes
    const int   primID = optixGetPrimitiveIndex();
    const uint3 index  = sbtData.index[primID];

    // get barycentric coordinates
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    // compute normal
    const float4 n
        = (1.f-u-v) * sbtData.vertexD.normal[index.x]
        +         u * sbtData.vertexD.normal[index.y]
        +         v * sbtData.vertexD.normal[index.z];
    // ray payload

    float3 normal = normalize(make_float3(n));

    // entering glass
    //if (dot(optixGetWorldRayDirection(), normal) < 0)

    float3 afterPRD = make_float3(1.0f);
    uint32_t u0, u1;
    packPointer( &afterPRD, u0, u1 );  

    const float3 pos = optixGetWorldRayOrigin() + optixGetRayTmax()*optixGetWorldRayDirection();
    //(1.f-u-v) * A + u * B + v * C;
    
    float3 rayDir = reflect(optixGetWorldRayDirection(), normal);
    optixTrace(optixLaunchParams.traversable,
        pos,
        rayDir,
        0.00001f,    // tmin
        1e20f,  // tmax
        0.0f,   // rayTime
        OptixVisibilityMask( 255 ),
        OPTIX_RAY_FLAG_NONE, //OPTIX_RAY_FLAG_NONE,
        PHONG,             // SBT offset
        RAY_TYPE_COUNT,     // SBT stride
        PHONG,             // missSBTIndex 
        u0, u1 );

    float3 &prd = *(float3*)getPRD<float3>();
    prd = make_float3(0.8,0.8,0.8) * afterPRD;
}




// -----------------------------------------------
// Glass Phong rays (Refração)

extern "C" __global__ void __closesthit__phong_glass() {

    const TriangleMeshSBTData &sbtData
      = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();  

    // retrieve primitive id and indexes
    const int   primID = optixGetPrimitiveIndex();
    const uint3 index  = sbtData.index[primID];

    // get barycentric coordinates
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    // compute normal
    const float4 n
        = (1.f-u-v) * sbtData.vertexD.normal[index.x]
        +         u * sbtData.vertexD.normal[index.y]
        +         v * sbtData.vertexD.normal[index.z];

    float3 normal = normalize(make_float3(n));
    const float3 normRayDir = optixGetWorldRayDirection();

    // new ray direction
    float3 rayDir;
    // entering glass
    float dotP;
    if (dot(normRayDir, normal) < 0) {
        dotP = dot(normRayDir, -normal);
        rayDir = refract(normRayDir, normal, 0.66);
    }
    // exiting glass
    else {
        dotP = 0;
        rayDir = refract(normRayDir, -normal, 1.5);
    }

    const float3 pos = optixGetWorldRayOrigin() + optixGetRayTmax() * optixGetWorldRayDirection();
    
    float3 refractPRD = make_float3(0.0f);
    uint32_t u0, u1;
    packPointer( &refractPRD, u0, u1 );  
    
    if (length(rayDir) > 0)
        optixTrace(optixLaunchParams.traversable,
            pos,
            rayDir,
            0.00001f,    // tmin
            1e20f,  // tmax
            0.0f,   // rayTime
            OptixVisibilityMask( 255 ),
            OPTIX_RAY_FLAG_NONE, //OPTIX_RAY_FLAG_NONE,
            PHONG,             // SBT offset
            RAY_TYPE_COUNT,     // SBT stride
            PHONG,             // missSBTIndex 
            u0, u1 );

    // ray payload 
    float3 &prd = *(float3*)getPRD<float3>();
 
    float3 reflectPRD = make_float3(0.0f);
    if (dotP > 0) {
        float3 reflectDir = reflect(normRayDir, normal);        
        packPointer( &reflectPRD, u0, u1 );  
        optixTrace(optixLaunchParams.traversable,
            pos,
            reflectDir,
            0.00001f,    // tmin
            1e20f,  // tmax
            0.0f,   // rayTime
            OptixVisibilityMask( 255 ),
            OPTIX_RAY_FLAG_NONE, //OPTIX_RAY_FLAG_NONE,
            PHONG,             // SBT offset
            RAY_TYPE_COUNT,     // SBT stride
            PHONG,             // missSBTIndex 
            u0, u1 );
        float r0 = (1.5f - 1.0f)/(1.5f + 1.0f);
        r0 = r0*r0 + (1-r0*r0) * pow(1-dotP,5);
        prd =  refractPRD * (1-r0) + r0*reflectPRD;
    }
    else
        prd =  refractPRD ;
}



extern "C" __global__ void __anyhit__phong_glass() {

}


// miss sets the background color
extern "C" __global__ void __miss__phong_glass() {

    float3 &prd = *(float3*)getPRD<float3>();
    // set blue as background color
    prd = make_float3(0.0f, 0.0f, 1.0f);
}


// -----------------------------------------------
// Glass Shadow rays

extern "C" __global__ void __closesthit__shadow_glass() {

    // ray payload
    float afterPRD = 1.0f;
    uint32_t u0, u1;
    packPointer( &afterPRD, u0, u1 );  

    const float3 pos = optixGetWorldRayOrigin() + optixGetRayTmax()*optixGetWorldRayDirection();
    
    // trace primary ray
    optixTrace(optixLaunchParams.traversable,
        pos,
        optixGetWorldRayDirection(),
        0.001f,    // tmin
        1e20f,  // tmax
        0.0f,   // rayTime
        OptixVisibilityMask( 255 ),
        OPTIX_RAY_FLAG_NONE, //OPTIX_RAY_FLAG_NONE,
        SHADOW,             // SBT offset
        RAY_TYPE_COUNT,     // SBT stride
        SHADOW,             // missSBTIndex 
        u0, u1 );

    float &prd = *(float*)getPRD<float>();
    prd = 0.95f * afterPRD;
}


// any hit for shadows
extern "C" __global__ void __anyhit__shadow_glass() {

}


// miss for shadows
extern "C" __global__ void __miss__shadow_glass() {

    float &prd = *(float*)getPRD<float>();
    // set blue as background color
    prd = 1.0f;
}





// -----------------------------------------------
// Primary Rays

extern "C" __global__ void __raygen__renderFrame() {

    const int ix = optixGetLaunchIndex().x;
    const int iy = optixGetLaunchIndex().y;
    const auto &camera = optixLaunchParams.camera;  

    const int &maxDepth = optixLaunchParams.frame.maxDepth;
 
    float squaredRaysPerPixel = float(optixLaunchParams.frame.raysPerPixel);
    float2 delta = make_float2(1.0f/squaredRaysPerPixel, 1.0f/squaredRaysPerPixel);

    float3 result = make_float3(0.0f);

    uint32_t seed = tea<4>( ix * optixGetLaunchDimensions().x + iy, optixLaunchParams.frame.frame );

    for (int i = 0; i < squaredRaysPerPixel; ++i) {
        for (int j = 0; j < squaredRaysPerPixel; ++j) {

            const float2 subpixel_jitter = make_float2( delta.x * (i + rnd(seed)), delta.y * (j + rnd( seed )));
            const float2 screen(make_float2(ix + subpixel_jitter.x, iy + subpixel_jitter.y)
                            / make_float2(optixGetLaunchDimensions().x, optixGetLaunchDimensions().y) * 2.0 - 1.0);
        
            // note: nau already takes into account the field of view and ratio when computing 
            // camera horizontal and vertical
            float3 origin = camera.position;
            float3 rayDir = normalize(camera.direction
                                + (screen.x ) * camera.horizontal
                                + (screen.y ) * camera.vertical);

            RadiancePRD prd;
            prd.emitted      = make_float3(0.f);
            prd.radiance     = make_float3(0.f);
            prd.attenuation  = make_float3(1.f);
            prd.countEmitted = true;
            prd.done         = false;
            prd.seed         = seed;

            uint32_t u0, u1;
            packPointer( &prd, u0, u1 );             
            
            for (int k = 0; k < maxDepth && !prd.done; ++k) {

                optixTrace(optixLaunchParams.traversable,
                        origin,
                        rayDir,
                        0.1f,    // tmin
                        50000.0f,  // tmax
                        0.0f, OptixVisibilityMask( 1 ),
                        OPTIX_RAY_FLAG_NONE, RAIDANCE, RAY_TYPE_COUNT, RAIDANCE, u0, u1 );

                result += prd.emitted;
                result += prd.radiance * prd.attenuation;

                origin = prd.origin;
                rayDir = prd.direction;
            }
        }
    }

    result = result / (squaredRaysPerPixel*squaredRaysPerPixel);
    float gamma = optixLaunchParams.global->gamma;
    // compute index
    const uint32_t fbIndex = ix + iy*optixGetLaunchDimensions().x;

    optixLaunchParams.global->accumBuffer[fbIndex] = 
        (optixLaunchParams.global->accumBuffer[fbIndex] * optixLaunchParams.frame.subFrame +
        make_float4(result.x, result.y, result.z, 1)) /(optixLaunchParams.frame.subFrame+1);

    
    float4 rgbaf = optixLaunchParams.global->accumBuffer[fbIndex];
    //convert float (0-1) to int (0-255)
    const int r = int(255.0f*min(1.0f, pow(rgbaf.x, 1/gamma)));
    const int g = int(255.0f*min(1.0f, pow(rgbaf.y, 1/gamma)));
    const int b = int(255.0f*min(1.0f, pow(rgbaf.z, 1/gamma))) ;

    // convert to 32-bit rgba value 
    const uint32_t rgba = 0xff000000 | (r<<0) | (g<<8) | (b<<16);
    // write to output buffer
    optixLaunchParams.frame.colorBuffer[fbIndex] = rgba;

    if (optixLaunchParams.frame.frame == 0 && ix == 0 && iy == 0) {

		// print info to console
		printf("===========================================\n");
        printf("Nau Ray-Tracing Debug\n");
        const float4 &ld = optixLaunchParams.global->lightPos;
        printf("LightPos: %f, %f %f %f\n", ld.x,ld.y,ld.z,ld.w);
        printf("Attenuation: %d\n", optixLaunchParams.global->attenuation);
        printf("Launch dim: %u %u\n", optixGetLaunchDimensions().x, optixGetLaunchDimensions().y);
        printf("Rays per pixel squared: %d \n", optixLaunchParams.frame.raysPerPixel);
        printf("Max Depth: %d \n", optixLaunchParams.global->maxDepth);
		printf("===========================================\n");
	}
}
  

