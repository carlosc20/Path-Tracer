#include "hip/hip_runtime.h"
#include "common.h"

// -----------------------------------------------
// Glossy Reflections (Specular materials with rugosity) 
extern "C" __global__ void __closesthit__glossy() {

    const TriangleMeshSBTData &sbtData
      = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();  

    RadiancePRD &prd = *(RadiancePRD *)getPRD<RadiancePRD>();

    // retrieve primitive id and indexes
    const int   primID = optixGetPrimitiveIndex();
    const uint3 index  = sbtData.index[primID];

    // get barycentric coordinates
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    // compute normal
    const float4 n
        = (1.f-u-v) * sbtData.vertexD.normal[index.x]
        +         u * sbtData.vertexD.normal[index.y]
        +         v * sbtData.vertexD.normal[index.z];

    float3 nn = normalize(make_float3(n));

    // intersection position
    const float3 &rayDir =  optixGetWorldRayDirection();
    const float3 pos = optixGetWorldRayOrigin() + optixGetRayTmax() * rayDir ;

    if (dot(nn, rayDir) > 0.0)
        nn = -nn;


    const float glossiness = optixLaunchParams.global->glossiness;

    float3 nextRayDir;
    float3 reflectDir = reflect(optixGetWorldRayDirection(), nn);
    uint32_t seed = prd.seed;

    do {
        const float z1 = rnd(seed);
        const float z2 = rnd(seed);
        cosine_power_sample_hemisphere( z1, z2, nextRayDir, glossiness );
        Onb onb( reflectDir );
        onb.inverse_transform( nextRayDir );
    } while (dot(nextRayDir, nn) < 0.001);
    prd.seed = seed;

    // set origin and direction for next ray
    prd.direction = nextRayDir;
    prd.origin    = pos;

    prd.attenuation *= sbtData.diffuse;
}