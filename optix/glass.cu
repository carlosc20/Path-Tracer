#include "hip/hip_runtime.h"
#include "common.h"

// -----------------------------------------------
// Glass Phong rays (Refração)
extern "C" __global__ void __closesthit__glass() {

    const TriangleMeshSBTData &sbtData
      = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();  

    RadiancePRD &prd = *(RadiancePRD *)getPRD<RadiancePRD>();

    // retrieve primitive id and indexes
    const int   primID = optixGetPrimitiveIndex();
    const uint3 index  = sbtData.index[primID];

    // get barycentric coordinates
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    // compute normal
    const float4 n
        = (1.f-u-v) * sbtData.vertexD.normal[index.x]
        +         u * sbtData.vertexD.normal[index.y]
        +         v * sbtData.vertexD.normal[index.z];

    float3 nn = normalize(make_float3(n));

    // intersection position
    const float3 &rayDir =  optixGetWorldRayDirection();
    const float3 pos = optixGetWorldRayOrigin() + optixGetRayTmax() * rayDir;

    // if (dot(nn, rayDir) > 0.0)
    //    nn = -nn;

    float3 nextRayDir;

    // entering glass
    float dotP;
    if (dot(rayDir, nn) < 0) {
        dotP = dot(rayDir, -nn);
        nextRayDir = refract(rayDir, nn, 0.66);
    }
    // exiting glass
    else {
        dotP = 0;
        nextRayDir = refract(rayDir, -nn, 1.5);
    }

    // didn't hit light
    prd.emitted = make_float3(0.0f);
    prd.countEmitted = false;

    if (length(nextRayDir) > 0) // why?
        prd.direction = nextRayDir;

    if (dotP > 0) {
        uint32_t seed = prd.seed;
        const float z = rnd(seed);
        prd.seed = seed;

        // refractive indices
        const float RI_AIR = 1.0f;
        const float RI_GLASS = 1.5f;

        // Reflection coefficient
        float r0 = (RI_GLASS - RI_AIR)/(RI_GLASS + RI_AIR);
        r0 = r0 * r0;
        // Schlick's approximation
        r0 = r0 + (1 - r0) * pow(1-dotP,5);

        // next ray has probability of being used for refraction or reflexion based on r0
        // aprox: refract * (1-r0) + reflect * r0;
        if(z < r0) {
            float3 reflectDir = reflect(rayDir, nn);        
            prd.direction = reflectDir;
        }
    }

    prd.origin = pos;

    // attenuation?
}


// -----------------------------------------------
// Glass Shadow rays
extern "C" __global__ void __closesthit__shadow_glass() {
    optixSetPayload_0( static_cast<uint32_t>(true));
}

/*
extern "C" __global__ void __closesthit__shadow_glass() {

    // ray payload
    float afterPRD = 1.0f;
    uint32_t u0, u1;
    packPointer( &afterPRD, u0, u1 );  

    // intersection position
    const float3 &rayDir =  optixGetWorldRayDirection();
    const float3 pos = optixGetWorldRayOrigin() + optixGetRayTmax() * rayDir;

    
    uint32_t occluded = 0u;
    optixTrace(optixLaunchParams.traversable,
        pos,
        rayDir,
        0.1f,                    // tmin
        1e20f,           // tmax
        0.0f,                    // rayTime
        OptixVisibilityMask( 255 ),
        OPTIX_RAY_FLAG_NONE,
        SHADOW,                 // SBT offset
        RAY_TYPE_COUNT,         // SBT stride
        SHADOW,                 // missSBTIndex
        occluded);

    // attenuation?
}
*/