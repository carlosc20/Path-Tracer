#include "hip/hip_runtime.h"
#include "common.h"

// -----------------------------------------------
// Glass Phong rays (Refração)

extern "C" __global__ void __closesthit__phong_glass() {

    const TriangleMeshSBTData &sbtData
      = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();  

    // retrieve primitive id and indexes
    const int   primID = optixGetPrimitiveIndex();
    const uint3 index  = sbtData.index[primID];

    // get barycentric coordinates
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    // compute normal
    const float4 n
        = (1.f-u-v) * sbtData.vertexD.normal[index.x]
        +         u * sbtData.vertexD.normal[index.y]
        +         v * sbtData.vertexD.normal[index.z];

    float3 normal = normalize(make_float3(n));
    const float3 normRayDir = optixGetWorldRayDirection();

    // new ray direction
    float3 rayDir;
    // entering glass
    float dotP;
    if (dot(normRayDir, normal) < 0) {
        dotP = dot(normRayDir, -normal);
        rayDir = refract(normRayDir, normal, 0.66);
    }
    // exiting glass
    else {
        dotP = 0;
        rayDir = refract(normRayDir, -normal, 1.5);
    }

    const float3 pos = optixGetWorldRayOrigin() + optixGetRayTmax() * optixGetWorldRayDirection();
    
    float3 refractPRD = make_float3(0.0f);
    uint32_t u0, u1;
    packPointer( &refractPRD, u0, u1 );  
    
    if (length(rayDir) > 0)
        optixTrace(optixLaunchParams.traversable,
            pos,
            rayDir,
            0.00001f,    // tmin
            1e20f,  // tmax
            0.0f,   // rayTime
            OptixVisibilityMask( 255 ),
            OPTIX_RAY_FLAG_NONE, //OPTIX_RAY_FLAG_NONE,
            PHONG,             // SBT offset
            RAY_TYPE_COUNT,     // SBT stride
            PHONG,             // missSBTIndex 
            u0, u1 );

    // ray payload 
    float3 &prd = *(float3*)getPRD<float3>();
 
    float3 reflectPRD = make_float3(0.0f);
    if (dotP > 0) {
        float3 reflectDir = reflect(normRayDir, normal);        
        packPointer( &reflectPRD, u0, u1 );  
        optixTrace(optixLaunchParams.traversable,
            pos,
            reflectDir,
            0.00001f,    // tmin
            1e20f,  // tmax
            0.0f,   // rayTime
            OptixVisibilityMask( 255 ),
            OPTIX_RAY_FLAG_NONE, //OPTIX_RAY_FLAG_NONE,
            PHONG,             // SBT offset
            RAY_TYPE_COUNT,     // SBT stride
            PHONG,             // missSBTIndex 
            u0, u1 );
        float r0 = (1.5f - 1.0f)/(1.5f + 1.0f);
        r0 = r0*r0 + (1-r0*r0) * pow(1-dotP,5);
        prd =  refractPRD * (1-r0) + r0*reflectPRD;
    }
    else
        prd =  refractPRD ;
}



extern "C" __global__ void __anyhit__phong_glass() {

}


// miss sets the background color
extern "C" __global__ void __miss__phong_glass() {

    float3 &prd = *(float3*)getPRD<float3>();
    // set blue as background color
    prd = make_float3(0.0f, 0.0f, 1.0f);
}


// -----------------------------------------------
// Glass Shadow rays

extern "C" __global__ void __closesthit__shadow_glass() {

    // ray payload
    float afterPRD = 1.0f;
    uint32_t u0, u1;
    packPointer( &afterPRD, u0, u1 );  

    const float3 pos = optixGetWorldRayOrigin() + optixGetRayTmax()*optixGetWorldRayDirection();
    
    // trace primary ray
    optixTrace(optixLaunchParams.traversable,
        pos,
        optixGetWorldRayDirection(),
        0.001f,    // tmin
        1e20f,  // tmax
        0.0f,   // rayTime
        OptixVisibilityMask( 255 ),
        OPTIX_RAY_FLAG_NONE, //OPTIX_RAY_FLAG_NONE,
        SHADOW,             // SBT offset
        RAY_TYPE_COUNT,     // SBT stride
        SHADOW,             // missSBTIndex 
        u0, u1 );

    float &prd = *(float*)getPRD<float>();
    prd = 0.95f * afterPRD;
}


// any hit for shadows
extern "C" __global__ void __anyhit__shadow_glass() {

}


// miss for shadows
extern "C" __global__ void __miss__shadow_glass() {

    float &prd = *(float*)getPRD<float>();
    // set blue as background color
    prd = 1.0f;
}


