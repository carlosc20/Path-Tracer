#include "hip/hip_runtime.h"
#include "common.h"

// -----------------------------------------------
// Glass Phong rays (Refração)
extern "C" __global__ void __closesthit__glass() {

    const TriangleMeshSBTData &sbtData
      = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();  

    RadiancePRD &prd = *(RadiancePRD *)getPRD<RadiancePRD>();

    // retrieve primitive id and indexes
    const int   primID = optixGetPrimitiveIndex();
    const uint3 index  = sbtData.index[primID];

    // get barycentric coordinates
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    // compute normal
    const float4 n
        = (1.f-u-v) * sbtData.vertexD.normal[index.x]
        +         u * sbtData.vertexD.normal[index.y]
        +         v * sbtData.vertexD.normal[index.z];

    float3 nn = normalize(make_float3(n));

    // intersection position
    const float3 &rayDir =  optixGetWorldRayDirection();
    const float3 pos = optixGetWorldRayOrigin() + optixGetRayTmax() * rayDir;

    // if (dot(nn, rayDir) > 0.0)
    //    nn = -nn;

    // refractive indices
    const float RI_AIR = 1.0f;
    const float refractionIndex = optixLaunchParams.global->refractionIndex; // glass -> 1.5
    
    float3 nextRayDir;

    // entering glass
    float dotP;
    if (dot(rayDir, nn) < 0) {
        dotP = dot(rayDir, -nn);
        nextRayDir = refract(rayDir, nn, RI_AIR/refractionIndex);
    }
    // exiting glass
    else {
        dotP = 0;
        nextRayDir = refract(rayDir, -nn, refractionIndex/RI_AIR);
    }


    if (length(nextRayDir) > 0) // why?
        prd.direction = nextRayDir;

    if (dotP > 0) {
        uint32_t seed = prd.seed;
        const float z = rnd(seed);
        prd.seed = seed;


        
        

        // Reflection coefficient
        float r0 = (refractionIndex - RI_AIR)/(refractionIndex + RI_AIR);
        r0 = r0 * r0;
        // Schlick's approximation
        r0 = r0 + (1 - r0) * pow(1-dotP,5);

        // next ray has probability of being used for refraction or reflexion based on r0
        // splitting: refract * (1-r0) + reflect * r0;
        if(z <= r0) {    
            prd.direction = reflect(rayDir, nn); 
        }
    }

    prd.origin = pos;

    prd.attenuation *= sbtData.diffuse;
}


// -----------------------------------------------
// Glass Shadow rays

extern "C" __global__ void __closesthit__shadow_glass() {
    optixSetPayload_0( static_cast<uint32_t>(true));
}