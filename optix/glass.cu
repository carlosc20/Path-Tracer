#include "hip/hip_runtime.h"
#include "common.h"

// -----------------------------------------------
// Glass Phong rays (Refração)
extern "C" __global__ void __closesthit__glass() {

    const TriangleMeshSBTData &sbtData
      = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();  

    RadiancePRD &prd = *(RadiancePRD *)getPRD<RadiancePRD>();

    // retrieve primitive id and indexes
    const int   primID = optixGetPrimitiveIndex();
    const uint3 index  = sbtData.index[primID];

    // get barycentric coordinates
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    // compute normal
    const float4 n
        = (1.f-u-v) * sbtData.vertexD.normal[index.x]
        +         u * sbtData.vertexD.normal[index.y]
        +         v * sbtData.vertexD.normal[index.z];

    float3 nn = normalize(make_float3(n));

    // intersection position
    const float3 &rayDir =  optixGetWorldRayDirection();
    const float3 pos = optixGetWorldRayOrigin() + optixGetRayTmax() * rayDir;

    // refractive indices
    const float RI_AIR = 1.0f;
    const float refractionIndex = optixLaunchParams.global->refractionIndex; // glass -> 1.5
    
    float3 nextRayDir;

    // entering glass
    float dotP;
    if (dot(rayDir, nn) < 0) {
        dotP = dot(rayDir, -nn);
        nextRayDir = refract(rayDir, nn, RI_AIR/refractionIndex);
    }
    // exiting glass
    else {
        dotP = 0;
        nextRayDir = refract(rayDir, -nn, refractionIndex/RI_AIR);
    }
    
    // ?
    if (length(nextRayDir) > 0)
        prd.direction = nextRayDir;


    // entering glass reflection/refraction
    if (dotP > 0) {
        // Reflection coefficient
        float r0 = (refractionIndex - RI_AIR)/(refractionIndex + RI_AIR);

        // Schlick's approximation
        r0 = r0 * r0;
        r0 = r0 + (1 - r0) * pow(1-dotP,5);

        // next ray has probability of being used for refraction or reflexion based on r0
        // splitting: refract * (1-r0) + reflect * r0;
        uint32_t seed = prd.seed;
        const float z = rnd(seed);
        prd.seed = seed;

        if(z <= r0) {
            prd.direction = reflect(rayDir, nn); 
        }
    }

    // set prd
    prd.origin = pos;

    prd.attenuation *= sbtData.diffuse;
    prd.specularBounce = true;
}