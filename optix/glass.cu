#include "hip/hip_runtime.h"
#include "common.h"

// -----------------------------------------------
// Glass Phong rays (Refração)
extern "C" __global__ void __closesthit__glass() {

    const TriangleMeshSBTData &sbtData
      = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();  

    RadiancePRD &prd = *(RadiancePRD *)getPRD<RadiancePRD>();

    // retrieve primitive id and indexes
    const int   primID = optixGetPrimitiveIndex();
    const uint3 index  = sbtData.index[primID];

    // get barycentric coordinates
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    // compute normal
    const float4 n
        = (1.f-u-v) * sbtData.vertexD.normal[index.x]
        +         u * sbtData.vertexD.normal[index.y]
        +         v * sbtData.vertexD.normal[index.z];

    float3 nn = normalize(make_float3(n));

    // intersection position
    const float3 &rayDir =  optixGetWorldRayDirection();
    const float3 pos = optixGetWorldRayOrigin() + optixGetRayTmax() * rayDir;

    // if (dot(nn, rayDir) > 0.0)
    //    nn = -nn;

    float3 nextRayDir;

    // entering glass
    float dotP;
    if (dot(rayDir, nn) < 0) {
        dotP = dot(rayDir, -nn);
        nextRayDir = refract(rayDir, nn, 0.66);
    }
    // exiting glass
    else {
        dotP = 0;
        nextRayDir = refract(rayDir, -nn, 1.5);
    }


    if (length(nextRayDir) > 0) // why?
        prd.direction = nextRayDir;

    if (dotP > 0) {
        uint32_t seed = prd.seed;
        const float z = rnd(seed);
        prd.seed = seed;

        // refractive indices
        const float RI_AIR = 1.0f;
        const float RI_GLASS = 1.5f;

        // Reflection coefficient
        float r0 = (RI_GLASS - RI_AIR)/(RI_GLASS + RI_AIR);
        r0 = r0 * r0;
        // Schlick's approximation
        r0 = r0 + (1 - r0) * pow(1-dotP,5);

        // next ray has probability of being used for refraction or reflexion based on r0
        // aprox: refract * (1-r0) + reflect * r0;
        if(z <= r0) {
            float3 reflectDir = reflect(rayDir, nn);        
            prd.direction = reflectDir;
        }
    }

    prd.origin = pos;

    // attenuation?
}


// -----------------------------------------------
// Glass Shadow rays

extern "C" __global__ void __closesthit__shadow_glass() {
    optixSetPayload_0( static_cast<uint32_t>(true));
}